#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define PROFILING 1
#ifdef PROFILING
#include "RDTimer.h"
#endif
#ifdef __cplusplus
extern "C" {
#endif

//========================================================================================================================================================================================================200
//	DEFINE/INCLUDE
//========================================================================================================================================================================================================200

//======================================================================================================================================================150
//	COMMON
//======================================================================================================================================================150

#include "../common.h"								// (in main program directory)			needed to recognized input variables

//======================================================================================================================================================150
//	UTILITIES
//======================================================================================================================================================150

#include "../util/cuda/hip/hip_runtime.h"					// (in path specified to compiler)	needed by for device functions
//#include "../util/timer/timer.h"					// (in path specified to compiler)	needed by timer

//======================================================================================================================================================150
//	KERNEL
//======================================================================================================================================================150

#include "./kernel_gpu_cuda.cu"						// (in current directory)	GPU kernel, cannot include with header file because of complications with passing of constant memory variables

//======================================================================================================================================================150
//	HEADER
//======================================================================================================================================================150

#include "./kernel_gpu_cuda_wrapper.h"				// (in current directory)

//========================================================================================================================================================================================================200
//	KERNEL_GPU_CUDA_WRAPPER FUNCTION
//========================================================================================================================================================================================================200


void 
kernel_gpu_cuda_wrapper(record *records,
						long records_mem,
						knode *knodes,
						long knodes_elem,
						long knodes_mem,

						int order,
						long maxheight,
						int count,

						long *currKnode,
						long *offset,
						int *keys,
						record *ans)
{

	//======================================================================================================================================================150
	//	CPU VARIABLES
	//======================================================================================================================================================150

	// timer
/*	long long time0;
	long long time1;
	long long time2;
	long long time3;
	long long time4;
	long long time5;
	long long time6;*/

//	time0 = get_time();

	//======================================================================================================================================================150
	//	GPU SETUP
	//======================================================================================================================================================150

	//====================================================================================================100
	//	INITIAL DRIVER OVERHEAD
	//====================================================================================================100

	hipDeviceSynchronize();

	//====================================================================================================100
	//	EXECUTION PARAMETERS
	//====================================================================================================100

	int numBlocks;
	numBlocks = count;									// max # of blocks can be 65,535
	int threadsPerBlock;
	threadsPerBlock = order < 1024 ? order : 1024;

	printf("# of blocks = %d, # of threads/block = %d (ensure that device can handle)\n", numBlocks, threadsPerBlock);

//	time1 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY				(MALLOC)
	//======================================================================================================================================================150

	//====================================================================================================100
	//	DEVICE IN
	//====================================================================================================100

	//==================================================50
	//	recordsD
	//==================================================50


/* malloc time-start */ 
#ifdef PROFILING
	float alloc_t1, cpu_to_gpu_t1,kernel_t1,gpu_to_cpu_t1;
        SimplePerfSerializer* serializeTime = new SimplePerfSerializer("./b+tree_1" );

        RDTimerCPU* rdtimercpu = new RDTimerCPU();

        rdtimercpu->Reset("Malloc Time");
        rdtimercpu->Start();
#endif

	record *recordsD;
	hipMalloc((void**)&recordsD, records_mem);
	checkCUDAError("hipMalloc  recordsD");

	//==================================================50
	//	knodesD
	//==================================================50

	knode *knodesD;
	hipMalloc((void**)&knodesD, knodes_mem);
	checkCUDAError("hipMalloc  recordsD");

	//==================================================50
	//	currKnodeD
	//==================================================50

	long *currKnodeD;
	hipMalloc((void**)&currKnodeD, count*sizeof(long));
	checkCUDAError("hipMalloc  currKnodeD");

	//==================================================50
	//	offsetD
	//==================================================50

	long *offsetD;
	hipMalloc((void**)&offsetD, count*sizeof(long));
	checkCUDAError("hipMalloc  offsetD");

	//==================================================50
	//	keysD
	//==================================================50

	int *keysD;
	hipMalloc((void**)&keysD, count*sizeof(int));
	checkCUDAError("hipMalloc  keysD");

	//====================================================================================================100
	//	DEVICE IN/OUT
	//====================================================================================================100

	//==================================================50
	//	ansD
	//==================================================50

	record *ansD;
	hipMalloc((void**)&ansD, count*sizeof(record));
	checkCUDAError("hipMalloc ansD");

//	time2 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY IN
	//====================================================================================================100

	//==================================================50
	//	recordsD
	//==================================================50

/* malloc time-stop, cpu-gpu transfer start */ 
#ifdef PROFILING
        alloc_t1 = rdtimercpu->Stop();
        serializeTime->Serialize(rdtimercpu);
        rdtimercpu->Reset("CPU to GPU Transfer Time");
        rdtimercpu->Start();
#endif

	hipMemcpy(recordsD, records, records_mem, hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy memD");

	//==================================================50
	//	knodesD
	//==================================================50

	hipMemcpy(knodesD, knodes, knodes_mem, hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy memD");

	//==================================================50
	//	currKnodeD
	//==================================================50

	hipMemcpy(currKnodeD, currKnode, count*sizeof(long), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy currKnodeD");

	//==================================================50
	//	offsetD
	//==================================================50

	hipMemcpy(offsetD, offset, count*sizeof(long), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy offsetD");

	//==================================================50
	//	keysD
	//==================================================50

	hipMemcpy(keysD, keys, count*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy keysD");

	//====================================================================================================100
	//	DEVICE IN/OUT
	//====================================================================================================100

	//==================================================50
	//	ansD
	//==================================================50

	hipMemcpy(ansD, ans, count*sizeof(record), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy ansD");

/*cpu-gpu transfer-stop, kernel exec-start */
#ifdef PROFILING
    cpu_to_gpu_t1 =  rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
    rdtimercpu->Reset("COMPUTE:Kernel Execution Time");
    hipDeviceSynchronize();
    rdtimercpu->Start();
#endif
//	time3 = get_time();

	//======================================================================================================================================================150
	// findK kernel
	//======================================================================================================================================================150

	hipLaunchKernelGGL(findK, dim3(numBlocks), dim3(threadsPerBlock), 0, 0, 	maxheight,

											knodesD,
											knodes_elem,

											recordsD,

											currKnodeD,
											offsetD,
											keysD,
											ansD);
	hipDeviceSynchronize();
	checkCUDAError("findK");

//	time4 = get_time();
/* kernel exec-stop, gpu-cpu transfer-start */
#ifdef PROFILING
    kernel_t1 = rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
    rdtimercpu->Reset("GPU to CPU Transfer Time");
    rdtimercpu->Start();
#endif


	//======================================================================================================================================================150
	//	GPU MEMORY			COPY (CONTD.)
	//======================================================================================================================================================150

	//====================================================================================================100
	//	DEVICE IN/OUT
	//====================================================================================================100

	//==================================================50
	//	ansD
	//==================================================50

	hipMemcpy(ans, ansD, count*sizeof(record), hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy ansD");

//	time5 = get_time();

/* gpu-cpu transfer- stop */ 
#ifdef PROFILING        
    gpu_to_cpu_t1= rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
#endif

	//======================================================================================================================================================150
	//	GPU MEMORY DEALLOCATION
	//======================================================================================================================================================150

	hipFree(recordsD);
	hipFree(knodesD);

	hipFree(currKnodeD);
	hipFree(offsetD);
	hipFree(keysD);
	hipFree(ansD);

//	time6 = get_time();

	//======================================================================================================================================================150
	//	DISPLAY TIMING
	//======================================================================================================================================================150

/*	printf("Time spent in different stages of GPU_CUDA KERNEL:\n");

	printf("%15.12f s, %15.12f % : GPU: SET DEVICE / DRIVER INIT\n",	(float) (time1-time0) / 1000000, (float) (time1-time0) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: ALO\n", 					(float) (time2-time1) / 1000000, (float) (time2-time1) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: COPY IN\n",					(float) (time3-time2) / 1000000, (float) (time3-time2) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU: KERNEL\n",						(float) (time4-time3) / 1000000, (float) (time4-time3) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU MEM: COPY OUT\n",				(float) (time5-time4) / 1000000, (float) (time5-time4) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: FRE\n", 					(float) (time6-time5) / 1000000, (float) (time6-time5) / (float) (time6-time0) * 100);

	printf("Total time:\n");
	printf("%.12f s\n", 												(float) (time6-time0) / 1000000);
*/

//========================================================================================================================================================================================================200
//	End
//========================================================================================================================================================================================================200

/* over all stop print all times delete timers */
#ifdef PROFILING    
      printf("time CPU to GPU memory copy = %lfs\n", cpu_to_gpu_t1);
      printf("time GPU to CPU memory copy back = %lfs\n", gpu_to_cpu_t1);
      printf("time GPU malloc = %lfs\n", alloc_t1);
      printf("time kernel = %lfs\n", kernel_t1);
          delete rdtimercpu;
          delete serializeTime;
 #endif

}

//========================================================================================================================================================================================================200
//	END
//========================================================================================================================================================================================================200

#ifdef __cplusplus
}
#endif
