#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#ifdef PROFILING
#include "RDTimer.h"
#endif

#define BLOCK_SIZE 256
#define STR_SIZE 256
#define DEVICE 0
#define HALO 1 // halo width along one direction when advancing to the next iteration

//#define BENCH_PRINT
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

int run(int argc, char** argv);

int rows, cols;
int* data;
int** wall;
int* result;
int* resultsRef;
#define M_SEED 9
int pyramid_height;

//#define BENCH_PRINT
#ifdef PROFILING
SimplePerfSerializer* serializeTime;
RDTimerCPU* rdtimercpu;
#endif

void
init(int argc, char** argv)
{
	if(argc==4){

		cols = atoi(argv[1]);

		rows = atoi(argv[2]);

                pyramid_height=atoi(argv[3]);
	}else{
                printf("Usage: dynproc row_len col_len pyramid_height\n");
                exit(0);
        }
	data = new int[rows*cols];

	wall = new int*[rows];

	for(int n=0; n<rows; n++)

		wall[n]=data+cols*n;

	result = new int[cols];
	resultsRef = new int[cols];
	

	int seed = M_SEED;

	srand(seed);



	for (int i = 0; i < rows; i++)

    {

        for (int j = 0; j < cols; j++)

        {

            wall[i][j] = rand() % 10;

        }

    }

	for (int j = 0; j < cols; j++)
        resultsRef[j] = wall[0][j];

#ifdef BENCH_PRINT

    for (int i = 0; i < rows; i++)

    {

        for (int j = 0; j < cols; j++)

        {

            printf("%d ",wall[i][j]) ;

        }

        printf("\n") ;

    }

#endif
}

void 
fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void dynproc_kernel( 
	int iteration,
	int *gpuWall,
	int *gpuSrc,
	int *gpuResults,
	int cols,
	int rows,
	int startStep,
	int border)
{

        __shared__ int prev[BLOCK_SIZE];
        __shared__ int result[BLOCK_SIZE];

	int bx = hipBlockIdx_x;
	int tx=hipThreadIdx_x;
	
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data

        // calculate the small block size
	int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkX = small_block_cols*bx-border;
        int blkXmax = blkX+BLOCK_SIZE-1;

        // calculate the global thread coordination
	int xidx = blkX+tx;
       
        // effective range within this block that falls within 
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

        int W = tx-1;
        int E = tx+1;
        
        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;

        bool isValid = IN_RANGE(tx, validXmin, validXmax);

	if(IN_RANGE(xidx, 0, cols-1)){
            prev[tx] = gpuSrc[xidx];
	}
	__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
        bool computed;
        for (int i=0; i<iteration ; i++){ 
            computed = false;
            if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
                  isValid){
                  computed = true;
                  int left = prev[W];
                  int up = prev[tx];
                  int right = prev[E];
                  int shortest = MIN(left, up);
                  shortest = MIN(shortest, right);
                  int index = cols*(startStep+i)+xidx;
                  result[tx] = shortest + gpuWall[index];
	
            }
            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)	 //Assign the computation range
                prev[tx]= result[tx];
	    __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
      }

      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
          gpuResults[xidx]=result[tx];		
      }
}
/*
   compute N time steps
*/
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols, \
	 int pyramid_height, int blockCols, int borderCols)
{
        dim3 dimBlock(BLOCK_SIZE);
        dim3 dimGrid(blockCols);  
	
        int src = 1, dst = 0;
	for (int t = 0; t < rows-1; t+=pyramid_height) {
            int temp = src;
            src = dst;
            dst = temp;
            hipLaunchKernelGGL(dynproc_kernel, dim3(dimGrid), dim3(dimBlock), 0, 0, 
                MIN(pyramid_height, rows-t-1), 
                gpuWall, gpuResult[src], gpuResult[dst],
                cols,rows, t, borderCols);
	}
        return dst;
}

int main(int argc, char** argv)
{
    int num_devices;
    
#ifdef PROFILING
	float overall_cpu_t;
    	RDTimerCPU* rdtimerOverallCpu = new RDTimerCPU();
    	rdtimerOverallCpu->Reset("Overall CPU Time");
    	serializeTime = new SimplePerfSerializer( argv[0] );
    	rdtimercpu = new RDTimerCPU();
    	rdtimerOverallCpu->Start();
#endif
    hipGetDeviceCount(&num_devices);
    if (num_devices > 1) hipSetDevice(DEVICE);

    int retVal= run(argc,argv);
    
#ifdef PROFILING    
	overall_cpu_t = rdtimerOverallCpu->Stop();
	serializeTime->Serialize(rdtimerOverallCpu);

	//printf("Overall CPU time = %fs\n", overall_cpu_t);
	
	delete rdtimercpu;
	delete rdtimerOverallCpu;
	delete serializeTime;
 #endif
    return retVal;
}

int run(int argc, char** argv)
{
	int retVal = 0;
    init(argc, argv);

    /* --------------- pyramid parameters --------------- */
    int borderCols = (pyramid_height)*HALO;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*HALO*2;
    int blockCols = cols/smallBlockCol+((cols%smallBlockCol==0)?0:1);

    printf("pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",\
	pyramid_height, cols, borderCols, BLOCK_SIZE, blockCols, smallBlockCol);
	
    int *gpuWall, *gpuResult[2];
    int size = rows*cols;
#ifdef PROFILING
	rdtimercpu->Reset("Malloc Time");
	rdtimercpu->Start();
#endif
    hipMalloc((void**)&gpuResult[0], sizeof(int)*cols);
    hipMalloc((void**)&gpuResult[1], sizeof(int)*cols);
    hipMalloc((void**)&gpuWall, sizeof(int)*(size-cols));
#ifdef PROFILING
    rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
    rdtimercpu->Reset("CPU to GPU Transfer Time");
    rdtimercpu->Start();
#endif
    hipMemcpy(gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice);
    
    hipMemcpy(gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice);
#ifdef PROFILING
    rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
    rdtimercpu->Reset("COMPUTE:Compute Time");
    rdtimercpu->Start();
#endif
    int final_ret = calc_path(gpuWall, gpuResult, rows, cols, \
	 pyramid_height, blockCols, borderCols);
#ifdef PROFILING
    rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
    rdtimercpu->Reset("GPU to CPU Transfer Time");
    rdtimercpu->Start();
#endif
    hipMemcpy(result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost);
#ifdef PROFILING    
    rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
#endif
	//
	int *srcRef, *dstRef, *tempRef;
    int minRef;

    dstRef = resultsRef;
    srcRef = new int[cols];

   // pin_stats_reset();
    for (int t = 0; t < rows-1; t++) {
        tempRef = srcRef;
        srcRef = dstRef;
        dstRef = tempRef;
        for(int n = 0; n < cols; n++){
          minRef = srcRef[n];
          if (n > 0)
            minRef = MIN(minRef, srcRef[n-1]);
          if (n < cols-1)
            minRef = MIN(minRef, srcRef[n+1]);
          dstRef[n] = wall[t+1][n]+minRef;
        }
    }

	for (int i = 0; i < cols; i++)
	{
		if (result[i] != dstRef[i])
		{
			printf("failed results at %d %d %d\n", i, result[i], dstRef[i]);
			retVal = 1;
		}
	}


	//

#ifdef BENCH_PRINT

    for (int i = 0; i < cols; i++)

            printf("%d ",data[i]) ;

    printf("\n") ;

    for (int i = 0; i < cols; i++)

            printf("%d ",result[i]) ;

    printf("\n") ;

#endif


    hipFree(gpuWall);
    hipFree(gpuResult[0]);
    hipFree(gpuResult[1]);

    delete [] data;
    delete [] wall;
    delete [] result;
	
	delete [] dstRef;
    delete [] srcRef;
	//delete [] resultsRef;
	return retVal;
}

