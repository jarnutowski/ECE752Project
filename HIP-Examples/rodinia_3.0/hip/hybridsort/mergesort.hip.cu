#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Includes
////////////////////////////////////////////////////////////////////////////////
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "mergesort.cuh"
#include "mergesort_kernel.hip.cu"
////////////////////////////////////////////////////////////////////////////////
// Defines
////////////////////////////////////////////////////////////////////////////////
#define BLOCKSIZE	256
#define ROW_LENGTH	BLOCKSIZE * 4
#define ROWS		4096

////////////////////////////////////////////////////////////////////////////////
// The mergesort algorithm
////////////////////////////////////////////////////////////////////////////////
float4* runMergeSort(int listsize, int divisions, 
				     float4 *d_origList, float4 *d_resultList, 
				     int *sizes, int *nullElements,
					 unsigned int *origOffsets)
{
	int *startaddr = (int *)malloc((divisions + 1)*sizeof(int)); 
	int largestSize = -1; 
	startaddr[0] = 0; 
	for(int i=1; i<=divisions; i++)
	{
		startaddr[i] = startaddr[i-1] + sizes[i-1];
		if(sizes[i-1] > largestSize) largestSize = sizes[i-1]; 
	}
	largestSize *= 4; 
//	printf("\n *** list size = %d *** \n",listsize);
	#ifdef USE_TEXTURES
	// Setup texture
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;
	/*#else
	float4 *dg_origList;
	hipMalloc((void**) &dg_origList, (listsize/4)*sizeof(float4));
        hipMemcpy(dg_origList,d_origList,(listsize/4)*sizeof(float4), hipMemcpyHostToDevice);*/
	#endif
	////////////////////////////////////////////////////////////////////////////
	// First sort all float4 elements internally
	////////////////////////////////////////////////////////////////////////////
	#ifdef MERGE_WG_SIZE_0
	const int THREADS = MERGE_WG_SIZE_0;
	#else
	const int THREADS = 256; 
	#endif
	dim3 threads(THREADS, 1);
	int blocks = ((listsize/4)%THREADS == 0) ? (listsize/4)/THREADS : (listsize/4)/THREADS + 1; 
	dim3 grid(blocks, 1);
	#ifdef USE_TEXTURES
	hipBindTexture(0,tex,d_origList, channelDesc, listsize*sizeof(float)); 
	hipLaunchKernelGGL(mergeSortFirst, dim3(grid), dim3(threads ), 0, 0, d_resultList, listsize);
	#else
	hipLaunchKernelGGL(mergeSortFirst, dim3(grid), dim3(threads ), 0, 0,d_origList, d_resultList, listsize);
	#endif
	////////////////////////////////////////////////////////////////////////////
	// Then, go level by level
	////////////////////////////////////////////////////////////////////////////
	#ifdef MEMCPYTOSYMBOL
	hipMemcpyToSymbol(HIP_SYMBOL(constStartAddr), startaddr, (divisions + 1)*sizeof(int)); 
	hipMemcpyToSymbol(HIP_SYMBOL(finalStartAddr), origOffsets, (divisions + 1)*sizeof(int)); 
	hipMemcpyToSymbol(HIP_SYMBOL(nullElems), nullElements, (divisions)*sizeof(int));
	#else
	int *dconstStartAddr ;
	unsigned int *dfinalStartAddr ;
	int *dnullElements ;
	hipMalloc((void**) &dconstStartAddr, (divisions + 1)*sizeof(int));
	hipMalloc((void**) &dfinalStartAddr, (divisions + 1)*sizeof(unsigned int));
	hipMalloc((void**) &dnullElements, (divisions)*sizeof(int));
	hipMemcpy(dconstStartAddr,startaddr,(divisions + 1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dfinalStartAddr,origOffsets,(divisions + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(dnullElements, nullElements, (divisions)*sizeof(int), hipMemcpyHostToDevice);
	#endif
	int nrElems = 2;
	while(true){
		int floatsperthread = (nrElems*4); 
		int threadsPerDiv = (int)ceil(largestSize/(float)floatsperthread); 
		int threadsNeeded = threadsPerDiv * divisions; 
		#ifdef MERGE_WG_SIZE_1
		threads.x = MERGE_WG_SIZE_1;
		#else
		threads.x = 208; 
		#endif
		grid.x = ((threadsNeeded%threads.x) == 0) ?
			threadsNeeded/threads.x : 
			(threadsNeeded/threads.x) + 1; 
		if(grid.x < 8){
			grid.x = 8; 
			threads.x = ((threadsNeeded%grid.x) == 0) ? 
				threadsNeeded / grid.x : 
				(threadsNeeded / grid.x) + 1; 
		}
		// Swap orig/result list
		float4 *tempList = d_origList; 
		d_origList = d_resultList; 
		d_resultList = tempList;
	/*	#ifdef USE_TEXTURES 
		hipBindTexture(0,tex, d_origList, channelDesc, listsize*sizeof(float));
                #ifdef MEMCPYTOSYMBOL
                hipLaunchKernel(reinterpret_cast<const void*>(mergeSortPass), dim3(grid), dim3(threads ), 0, 0, d_resultList, nrElems, threadsPerDiv);
                #else
                hipLaunchKernel(reinterpret_cast<const void*>(mergeSortPass), dim3(grid), dim3(threads ), 0, 0,d_resultList, nrElems, threadsPerDiv,startaddr);
                #endif
                #endif */

		#ifdef MEMCPYTOSYMBOL
		hipLaunchKernelGGL(mergeSortPass, dim3(grid), dim3(threads ), 0, 0,d_origList, d_resultList, nrElems, threadsPerDiv);
		#else
		hipLaunchKernelGGL(mergeSortPass, dim3(grid), dim3(threads ), 0, 0,d_origList, d_resultList, nrElems, threadsPerDiv,dconstStartAddr); 
		#endif
		nrElems *= 2; 
		floatsperthread = (nrElems*4); 
		if(threadsPerDiv == 1) break; 
	}
	////////////////////////////////////////////////////////////////////////////
	// Now, get rid of the NULL elements
	////////////////////////////////////////////////////////////////////////////
	#ifdef MERGE_WG_SIZE_0
	threads.x = MERGE_WG_SIZE_0;
	#else
	threads.x = 256; 
	#endif
	grid.x = ((largestSize%threads.x) == 0) ?
			largestSize/threads.x : 
			(largestSize/threads.x) + 1; 
	grid.y = divisions;
	#ifdef MEMCPYTOSYMBOL 
	hipLaunchKernelGGL(mergepack, dim3(grid), dim3(threads ), 0, 0, (float *)d_resultList, (float *)d_origList);
	#else
	hipLaunchKernelGGL(mergepack, dim3(grid), dim3(threads ), 0, 0, (float *)d_resultList, (float *)d_origList,dconstStartAddr,dfinalStartAddr,dnullElements);
	#endif
	free(startaddr);
	return d_origList; 
}
