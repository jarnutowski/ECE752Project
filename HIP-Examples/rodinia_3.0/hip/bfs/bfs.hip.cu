#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

  Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Created by Pawan Harish.
 ************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#ifdef __CUDA_CC__
#include "hip/hip_runtime.h"
#endif
#include <sys/time.h>

#define MAX_THREADS_PER_BLOCK 512

#define PROFILING 1
#ifdef PROFILING
#include "RDTimer.h"
#endif

int no_of_nodes;
int edge_list_size;
FILE *fp;

//Structure to hold a node information
struct Node
{
	int starting;
	int no_of_edges;
};

#include "kernel.hip.cu"
#include "kernel2.hip.cu"

void BFSGraph(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	no_of_nodes=0;
	edge_list_size=0;
	BFSGraph( argc, argv);
}

void Usage(int argc, char**argv){

fprintf(stderr,"Usage: %s <input_file>\n", argv[0]);

}
////////////////////////////////////////////////////////////////////////////////
//Apply BFS on a Graph using CUDA
////////////////////////////////////////////////////////////////////////////////
void BFSGraph( int argc, char** argv) 
{
	
    char *input_f;
	if(argc!=2){
	Usage(argc, argv);
	exit(0);
	}
	
	struct timeval start_time, end_time,start_time_kernel, end_time_kernel;
/* overall time - start */
#ifdef PROFILING
	float alloc_t, cpu_to_gpu_t,kernel_t,gpu_to_cpu_t,overall_cpu_t;
        RDTimerCPU* rdtimerOverallCpu = new RDTimerCPU();
        rdtimerOverallCpu->Reset("Overall CPU Time");
        rdtimerOverallCpu->Start();
#endif	
	input_f = argv[1];
	printf("Reading File\n");
	//Read in Graph from a file
	fp = fopen(input_f,"r");
	if(!fp)
	{
		printf("Error Reading graph file\n");
		return;
	}

	int source = 0;

	fscanf(fp,"%d",&no_of_nodes);

	int num_of_blocks = 1;
	int num_of_threads_per_block = no_of_nodes;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(no_of_nodes>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

	// allocate host memory
	Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
	bool *h_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
	bool *h_updating_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
	bool *h_graph_visited = (bool*) malloc(sizeof(bool)*no_of_nodes);

	int start, edgeno;   
	// initalize the memory
	for( unsigned int i = 0; i < no_of_nodes; i++) 
	{
		fscanf(fp,"%d %d",&start,&edgeno);
		h_graph_nodes[i].starting = start;
		h_graph_nodes[i].no_of_edges = edgeno;
		h_graph_mask[i]=false;
		h_updating_graph_mask[i]=false;
		h_graph_visited[i]=false;
	}

	//read the source node from the file
	fscanf(fp,"%d",&source);
	source=0;

	//set the source node as true in the mask
	h_graph_mask[source]=true;
	h_graph_visited[source]=true;

	fscanf(fp,"%d",&edge_list_size);

	int id,cost;
	int* h_graph_edges = (int*) malloc(sizeof(int)*edge_list_size);
	for(int i=0; i < edge_list_size ; i++)
	{
		fscanf(fp,"%d",&id);
		fscanf(fp,"%d",&cost);
		h_graph_edges[i] = id;
	}

	if(fp)
		fclose(fp);    

	printf("Read File\n");

        // allocate mem for the result on host side
        int* h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
        for(int i=0;i<no_of_nodes;i++)
                h_cost[i]=-1;
        h_cost[source]=0;
	
	/* start time */
	gettimeofday(&start_time, NULL);

/* malloc time-start */ 
#ifdef PROFILING
        SimplePerfSerializer* serializeTime = new SimplePerfSerializer( argv[0] );

        RDTimerCPU* rdtimercpu = new RDTimerCPU();

        rdtimercpu->Reset("Malloc Time");
        rdtimercpu->Start();
#endif	
	//Copy the Node list to device memory
	Node* d_graph_nodes;
	hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;

	//Copy the Edge List to device Memory
	int* d_graph_edges;
	hipMalloc( (void**) &d_graph_edges, sizeof(int)*edge_list_size) ;
	
	//Copy the Mask to device memory
        bool* d_graph_mask;
	hipMalloc( (void**) &d_graph_mask, sizeof(bool)*no_of_nodes) ;

	bool* d_updating_graph_mask;
	hipMalloc( (void**) &d_updating_graph_mask, sizeof(bool)*no_of_nodes) ;

	//Copy the Visited nodes array to device memory
	bool* d_graph_visited;
	hipMalloc( (void**) &d_graph_visited, sizeof(bool)*no_of_nodes) ;

	// allocate device memory for result
	int* d_cost;
	hipMalloc( (void**) &d_cost, sizeof(int)*no_of_nodes);

	//make a bool to check if the execution is over
	bool *d_over;
	hipMalloc( (void**) &d_over, sizeof(bool));

// copying to GPU

/* malloc time-stop, cpu-gpu transfer start */ 
#ifdef PROFILING
        alloc_t = rdtimercpu->Stop();
        serializeTime->Serialize(rdtimercpu);
        rdtimercpu->Reset("CPU to GPU Transfer Time");
        rdtimercpu->Start();
#endif
 
// nodelist
	hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;
//edgelist	
	hipMemcpy( d_graph_edges, h_graph_edges, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;
//mask
	hipMemcpy( d_graph_mask, h_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;
	hipMemcpy( d_updating_graph_mask, h_updating_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;
// visited nodes 
	hipMemcpy( d_graph_visited, h_graph_visited, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;
// device memory for result 
	hipMemcpy( d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) ;

	printf("Copied Everything to GPU memory\n");

	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	int k=0;
	printf("Start traversing the tree\n");
	bool stop;
	//Call the Kernel untill all the elements of Frontier are not false
	
	/* start time */
	gettimeofday(&start_time_kernel, NULL);
/*cpu-gpu transfer-stop, kernel exec-start */
#ifdef PROFILING
    cpu_to_gpu_t =  rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
    rdtimercpu->Reset("COMPUTE:Kernel Execution Time");
    //hipDeviceSynchronize();
    rdtimercpu->Start();
#endif
	
	do
	{
		//if no thread changes this value then the loop stops
		stop=false;
		hipMemcpy( d_over, &stop, sizeof(bool), hipMemcpyHostToDevice) ;
		hipLaunchKernelGGL(Kernel, dim3(grid), dim3(threads ), 0, 0,  d_graph_nodes, d_graph_edges, d_graph_mask, d_updating_graph_mask, d_graph_visited, d_cost, no_of_nodes);
		// check if kernel execution generated and error
		

		hipLaunchKernelGGL(Kernel2, dim3(grid), dim3(threads ), 0, 0,  d_graph_mask, d_updating_graph_mask, d_graph_visited, d_over, no_of_nodes);
		// check if kernel execution generated and error
		

		hipMemcpy( &stop, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
		k++;
	}
	while(stop);

/* kernel exec-stop, gpu-cpu transfer-start */
#ifdef PROFILING
    kernel_t = rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
    rdtimercpu->Reset("GPU to CPU Transfer Time");
    rdtimercpu->Start();
#endif
        /* end time */
        gettimeofday(&end_time_kernel, NULL);

	printf("Kernel Executed %d times\n",k);

	// copy result from device to host
	hipMemcpy( h_cost, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) ;
        
/* gpu-cpu transfer- stop */ 
#ifdef PROFILING        
    gpu_to_cpu_t= rdtimercpu->Stop();
    serializeTime->Serialize(rdtimercpu);
#endif
	/* end time */
        gettimeofday(&end_time, NULL);

	//Store the result into a file
	FILE *fpo = fopen("result.txt","w");
	for(int i=0;i<no_of_nodes;i++)
		fprintf(fpo,"%d) cost:%d\n",i,h_cost[i]);
	fclose(fpo);
	printf("Result stored in result.txt\n");

	/* printing runtime including mem alloc and copying */
/*	printf("Runtime on GPU including memory allocations: %.4lf s\n", end_time.tv_sec + end_time.tv_usec / 1000000.0 - start_time.tv_sec - start_time.tv_usec / 1000000.0);*/

	/* printing runtime of kernel execution */
       /* printf("Runtime on GPU for kernel execution: %.4lf s\n", end_time_kernel.tv_sec + end_time_kernel.tv_usec / 1000000.0 - start_time_kernel.tv_sec - start_time_kernel.tv_usec / 1000000.0);*/	
	
/* over all stop print all times delete timers */
#ifdef PROFILING    
    overall_cpu_t =  rdtimerOverallCpu->Stop();
    serializeTime->Serialize(rdtimerOverallCpu);
      printf("time CPU to GPU memory copy = %lfs\n", cpu_to_gpu_t);
      printf("time GPU to CPU memory copy back = %lfs\n", gpu_to_cpu_t);
      printf("time GPU malloc = %lfs\n", alloc_t);
      printf("time kernel = %lfs\n", kernel_t);
      printf("Overall CPU time = %lfs\n", overall_cpu_t);
      
          delete rdtimercpu;
          delete serializeTime;
          delete rdtimerOverallCpu;

 #endif
	// cleanup memory
	free( h_graph_nodes);
	free( h_graph_edges);
	free( h_graph_mask);
	free( h_updating_graph_mask);
	free( h_graph_visited);
	free( h_cost);
	hipFree(d_graph_nodes);
	hipFree(d_graph_edges);
	hipFree(d_graph_mask);
	hipFree(d_updating_graph_mask);
	hipFree(d_graph_visited);
	hipFree(d_cost);
}
