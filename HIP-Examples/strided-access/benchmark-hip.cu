#include "hip/hip_runtime.h"
//
// CUDA benchmark for measuring effective memory bandwidth for strided array access
//
// Author: Karl Rupp,  me@karlrupp.net
// License: MIT/X11 license, see file LICENSE.txt
//

#include <iostream>
#include <string>
#include <vector>
#include <sstream>
#include <stdexcept>

#include "hip/hip_runtime.h"

#include "benchmark-utils.hpp"


inline void cuda_last_error_check()
{
  hipError_t error_code = hipGetLastError();

  if (hipSuccess != error_code)
  {
    std::stringstream ss;
    ss << "CUDA Runtime API error " << error_code << ": " << hipGetErrorString( error_code ) << std::endl;
    throw std::runtime_error(ss.str());
  }
}


// Kernel for the benchmark
template<typename NumericT>
__global__ void elementwise_add(hipLaunchParm lp,
                                const NumericT * x,
                                const NumericT * y,
                                      NumericT * z,
                                unsigned int stride,
                                unsigned int size)
{
  for (unsigned int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
                    i < size;
                    i += hipGridDim_x * hipBlockDim_x)
    z[i*stride] = x[i*stride] + y[i*stride];
}


int main(int argc, char **argv)
{
  typedef float       NumericT;

  hipDeviceProp_t prop;
  hipError_t err = hipGetDeviceProperties(&prop, 0); if (err != hipSuccess) throw std::runtime_error("Failed to get CUDA device name");
  std::cout << "# Using device: " << prop.name << std::endl;

  // Set up work vectors
  std::size_t N =  1000000;

  std::vector<NumericT> host_x(32*N);
  NumericT *x, *y, *z;

  err = hipMalloc(&x, sizeof(NumericT) * 32 * N); if (err != hipSuccess) throw std::runtime_error("Failed to allocate CUDA memory for x");
  err = hipMalloc(&y, sizeof(NumericT) * 32 * N); if (err != hipSuccess) throw std::runtime_error("Failed to allocate CUDA memory for y");
  err = hipMalloc(&z, sizeof(NumericT) * 32 * N); if (err != hipSuccess) throw std::runtime_error("Failed to allocate CUDA memory for z");


  // Warmup calculation:
  hipLaunchKernel(reinterpret_cast<const void*>(elementwise_add), dim3(256), dim3(256), 0, 0, x, y, z,
                                static_cast<unsigned int>(1),
                                static_cast<unsigned int>(N));
  cuda_last_error_check();

  // Benchmark runs
  Timer timer;
  std::cout << "# stride     time       GB/sec" << std::endl;
  for (std::size_t stride = 1; stride <= 32 ; ++stride)
  {
    hipDeviceSynchronize();
    timer.start();

    // repeat calculation several times, then average
    for (std::size_t num_runs = 0; num_runs < 20; ++num_runs)
    {
      hipLaunchKernel(reinterpret_cast<const void*>(elementwise_add), dim3(256), dim3(256), 0, 0, x, y, z,
                                    static_cast<unsigned int>(stride),
                                    static_cast<unsigned int>(N));
      cuda_last_error_check();
    }
    hipDeviceSynchronize();
    double exec_time = timer.get();

    std::cout << "   " << stride << "        " << exec_time << "        " << 20 * 3.0 * sizeof(NumericT) * N / exec_time * 1e-9 << std::endl;
  }

  return EXIT_SUCCESS;
}

